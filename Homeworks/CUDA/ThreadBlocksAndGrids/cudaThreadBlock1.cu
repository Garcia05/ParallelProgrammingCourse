#include "hip/hip_runtime.h"
// program name: cudaThreadBlock.cu
// this program is designed for showing thread block example.
// author: Shane Cook (Nvidia .Inc) 
// modified by Yang Yang @ Peking University July 2017
// 
//
// Sketch diagram for thread blocks:
// -----------------------------------------------------------------------------------
// | thread block 0    || thread block 0   || thread block 1   || thread block 1    ||
// | thread bundle 0   || thread bundle 1  || thread bundle 0  || thread bundle 1   ||
// |  (thread 0~31)    || (thread 32~63)   || (thread 64~95)   || (thread 96~127)   ||
// -----------------------------------------------------------------------------------
//
// Adress space:
// -----------------------------------------------------------------------------------
// |                   ||                  ||                  ||                   ||
// |     adress        ||      adress      ||      adress      ||      adress       ||
// |     (0~31)        ||      (32~63)     ||      (64~95)     ||      (96~127)     ||
// -----------------------------------------------------------------------------------


// head files
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>




/* Cuda Kernel function: waht is my id */
__global__ void what_is_my_id(unsigned int * const block,
                              unsigned int * const thread,
                              unsigned int * const warp,
                              unsigned int * const calc_thread)
{
    /* Thread id equals to block index * block size + thread offset into the block */
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    block[thread_idx] = blockIdx.x;
    thread[thread_idx] = threadIdx.x;

    /* Calculate warp using built in variable warpSize */
    warp[thread_idx] = threadIdx.x / warpSize;

    calc_thread[thread_idx] = thread_idx;
}





#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * ARRAY_SIZE)

/* Declare statically four arrays of ARRAY_SIZE each */

unsigned int cpu_block[ARRAY_SIZE];
unsigned int cpu_thread[ARRAY_SIZE];
unsigned int cpu_warp[ARRAY_SIZE];
unsigned int cpu_calc_thread[ARRAY_SIZE];

int main(void)
{
    /* Total thread count = 2 * 64 = 128 */
    const unsigned int num_blocks = 2;
    const unsigned int num_threads = 64;
    char ch;

    /* Decalre pointers for GPU based params */
    unsigned int * gpu_block;
    unsigned int * gpu_thread;
    unsigned int * gpu_warp;
    unsigned int * gpu_calc_thread;

    /* Declare loop counter for use later */
    unsigned int i;

    /* Allocate four arrays on the GPU */
    hipMalloc((void **)&gpu_block, ARRAY_SIZE_IN_BYTES);         // Why here type is (void **)?
    hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_warp, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);

    /* Execute our cuda kernel */
    what_is_my_id<<<num_blocks, num_threads>>>(gpu_block, gpu_thread, gpu_warp, gpu_calc_thread);

    /* Copy back the gpu results to the CPU, from display RAM to RAM in physical*/
    hipMemcpy(cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_warp, gpu_warp, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

    /* Free the arrays on the GPU as now we're done with them */
    hipFree(gpu_block);
    hipFree(gpu_thread);
    hipFree(gpu_warp);
    hipFree(gpu_calc_thread);

    /* Iterate through the arrays and print */
    for (i = 0; i < ARRAY_SIZE; i++)
    {  
       if(i>0)
       { 
         if(cpu_warp[i] == 1 && cpu_warp[i-1] == 0) printf("\n");
         if(cpu_block[i] == 1 && cpu_block[i-1] ==0) printf("\n\n");
       } 
       printf("Calculated Thread: %3u - Block: %2u - Warp %2u - Thread %3u\n",
               cpu_calc_thread[i], cpu_block[i], cpu_warp[i], cpu_thread[i]);       
    }

    /* To avoid program exit automatically */
    ch = getchar();
}