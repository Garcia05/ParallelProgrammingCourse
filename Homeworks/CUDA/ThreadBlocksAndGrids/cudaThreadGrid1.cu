// program name: cudaThreadGrid.cu
// this program is designed for showing thread grid example.
// author: Shane Cook (Nvidia .Inc) 
// modified by Yang Yang @ Peking University July 2017
// 
//
// built in variables:
// gridDim.x -- number of thread blocks in X dim of thread grid 
// gridDim.y -- number of thread blocks in Y dim of thread grid
//
// blockDim.x -- number of threads in X dim of thread block
// blockDim.y -- number of threads in Y dim of thread block
//
// threadIdx.x -- thread index in X dim of thread block
// threadIdx.y -- thread index in Y dim of thread block
//
//
// Sketch diagram for thread grid for an array mapping:
//  o------> X
//  |
//  |
//  V Y
// ---------------------------------------------------------------------------------------------------- ---  ---     ---
// | array element 0  || array element 1  || array elemnt 2   || array element 3  || array element 4  |  ^    ^       ^
// |     X = 0        ||      X = 1       ||     X = 2        ||      X = 3       ||      X = 4       |  |    |       |
// |     Y = 0        ||      Y = 0       ||     Y = 0        ||      Y = 0       ||      Y = 0       |  |    V       |
// ---------------------------------------------------------------------------------------------------|  |   ---      |
// | array element 5  || array element 6  || array element 7  || array element 8  || array element 9  |  |blockDim.y  |
// |     X = 0        ||      X = 1       ||     X = 2        ||      X = 3       ||      X = 4       |  |            V
// |     Y = 1        ||      Y = 1       ||     Y = 1        ||      Y = 1       ||      Y = 1       |  |           ---
// ----------------------------------------------------------------------------------------------------  |         threadIdx.y
// | array element 10 || array element 11 || array element 12 || array element 13 || array element 14 |  |
// |     X = 0        ||      X = 1       ||     X = 2        ||      X = 3       ||      X = 4       |  |
// |     Y = 1        ||      Y = 1       ||     Y = 1        ||      Y = 1       ||      Y = 1       |  v
// ---------------------------------------------------------------------------------------------------- --- gridDim.y
//|<--------------------------------------------(gridDim.x)------------------------------------------>|
//|<---(blockDim.x)-->|
//|<------>| threadIdx.x

/*--------------------------------------------------------------------------------------------------------*/
// head files

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#include <conio.h>




/* Cuda Kernel function: waht is my id */
__global__ void what_is_my_id_2d_A(unsigned int * const block_x,
                                   unsigned int * const block_y,
                                   unsigned int * const thread,
                                   unsigned int * const calc_thread,
                                   unsigned int * const x_thread,
                                   unsigned int * const y_thread,
                                   unsigned int * const grid_dimx,
                                   unsigned int * const grid_dimy,
                                   unsigned int * const block_dimx,
                                   unsigned int * const block_dimy)
{
    /* Thread absolute id and id in X dim and Y dim */
    const unsigned int idx        = (blockIdx.x * blockDim.x) + threadIdx.x; 
    const unsigned int idy        = (blockIdx.y * blockDim.y) + threadIdx.y;
    const unsigned int thread_idx = ((gridDim.x * blockDim.x) * idy) + idx;

    block_x[thread_idx] = blockIdx.x;
    block_y[thread_idx] = blockIdx.y;
    thread[thread_idx] = threadIdx.x;
    calc_thread[thread_idx] = thread_idx;
    x_thread[thread_idx] = idx;
    y_thread[thread_idx] = idy;
    grid_dimx[thread_idx] = gridDim.x;
    grid_dimy[thread_idx] = gridDim.y;
    block_dimx[thread_idx] = blockDim.x;
    block_dimy[thread_idx] = blockDim.y;
}





/* Macro definition */
#define ARRAY_SIZE_X 32
#define ARRAY_SIZE_Y 16
#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X) * (ARRAY_SIZE_Y) * (sizeof(unsigned int)))

/* Declare statically four arrays of ARRAY_SIZE each */
unsigned int cpu_block_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_warp[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];




/* The main function */
int main(void)
{
    /* Total thread count = 32 * 4 = 128 */
    const dim3 threads_rect(32, 4);  /* 32 * 4 */
    const dim3 blocks_rect(1, 4);  
    
    /* Total thread count = 16 * 8 = 128 */
    const dim3 threads_square(16, 8);
    const dim3 blocks_square(2, 2);

    /* program pause wait for a getchar() in C++ */
    char ch;

    /* Decalre pointers for GPU based params */
    unsigned int * gpu_block_x;
    unsigned int * gpu_block_y;
    unsigned int * gpu_thread;
    unsigned int * gpu_warp;
    unsigned int * gpu_calc_thread;
    unsigned int * gpu_xthread;
    unsigned int * gpu_ythread;
    unsigned int * gpu_grid_dimx;
    unsigned int * gpu_grid_dimy;
    unsigned int * gpu_block_dimx;
    unsigned int * gpu_block_dimy;

    /* Allocate four arrays on the GPU */
    hipMalloc((void **)&gpu_block_x, ARRAY_SIZE_IN_BYTES);         // Why here type is (void **)?
    hipMalloc((void **)&gpu_block_y, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_xthread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_ythread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_grid_dimx, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_grid_dimy, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_dimx, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_dimy, ARRAY_SIZE_IN_BYTES);

    /* Execute our cuda kernel */
    for (int kernel = 0; kernel < 2 ; kernel ++)
    {
      switch (kernel)
      { 
          case 0:
          {
            /* Excute our kernel function */    
            what_is_my_id_2d_A<<<blocks_rect, threads_rect>>>(gpu_block_x, gpu_block_y, gpu_thread, gpu_calc_thread, 
            gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_grid_dimy, gpu_block_dimx, gpu_block_dimy);
          } break;

          case 1:
          {
            what_is_my_id_2d_A<<<blocks_square, threads_square>>>(gpu_block_x, gpu_block_y, gpu_thread, gpu_calc_thread, 
            gpu_xthread, gpu_ythread, gpu_grid_dimx, gpu_grid_dimy, gpu_block_dimx, gpu_block_dimy);
          } break;

          default: exit(1); break;
      }

      /* Copy back the gpu results to the CPU, from display RAM to RAM in physical */
      hipMemcpy(cpu_block_x, gpu_block_x, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);         
      hipMemcpy(cpu_block_y, gpu_block_y, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
      hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
      hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
      hipMemcpy(cpu_xthread, gpu_xthread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
      hipMemcpy(cpu_ythread, gpu_ythread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
      hipMemcpy(cpu_grid_dimx, gpu_grid_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
      hipMemcpy(cpu_grid_dimy, gpu_grid_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
      hipMemcpy(cpu_block_dimx, gpu_block_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
      hipMemcpy(cpu_block_dimy, gpu_block_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

      printf("\nKernel %d\n", kernel);
       
      /* Iterate through the arrays and print */
      for (int y = 0; y < ARRAY_SIZE_Y; y++)
      {
        for (int x = 0; x < ARRAY_SIZE_Y; x++)
        {
           printf("CT: %2u BKX: %1u BKY: %1u TID: %2u YTID: %2u XTID: %2u GDX: %1u GDY %1u BDX %1u BDY %1u\n",
                  cpu_calc_thread[y][x], cpu_block_x[y][x], cpu_block_y[y][x], cpu_thread[y][x], cpu_ythread[y][x],
                  cpu_xthread[y][x], cpu_grid_dimx[y][x], cpu_grid_dimx[y][x], cpu_block_dimx[y][x], cpu_block_dimy[y][x]);

           /* program pause and wait for a keyboard input */
           ch = getchar(); 
        }
      }
      /* waiting for any key so we can see the console window */
      printf("Press any key to continue\n");
      ch = getchar();

   }

      /* Free the arrays on the GPU as now we're done with them */
      hipFree(gpu_block_x);
      hipFree(gpu_block_y);
      hipFree(gpu_thread);
      hipFree(gpu_calc_thread);
      hipFree(gpu_xthread);
      hipFree(gpu_ythread);
      hipFree(gpu_grid_dimx);
      hipFree(gpu_grid_dimy);
      hipFree(gpu_block_dimx);
      hipFree(gpu_block_dimy);

      /* To avoid program exit automatically */
      ch = getchar();
}